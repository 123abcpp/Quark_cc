#include <assert.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

int main(void)
{
   
    // CUdevice device;
    // // cuInit(0);
    // cuDeviceGet(&device, 0);
    // printf("device name is %d",device);

    unsigned flags;
    int is_active;
    hipSetDevice(0);
    hipError_t status = hipDevicePrimaryCtxGetState(0, &flags, &is_active);
    if (status != hipSuccess) { 
        printf("got error cuevicePrimaryCTX \n");
     }
     printf("status is %d",status);
  
}